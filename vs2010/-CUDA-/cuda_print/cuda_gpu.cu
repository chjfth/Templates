#include <stdio.h>
#include <stdlib.h>
#include <tchar.h>

// Note: nvcc.exe will include necessary CUDA .h headers automatically,
// so we do not have to #include them manually.

extern"C" void test_call_cuda()
{
	int gpu_count = 0;
	hipError_t err = hipGetDeviceCount(&gpu_count);
	if(err)
	{
		_tprintf(_T("hipGetDeviceCount() error = %d\n"), err);
		return;
	}
	else
	{
		_tprintf(_T("gpu_count = %d\n"), gpu_count);
	}
}
