
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Note: nvcc.exe will include necessary CUDA .h headers automatically,
// so we do not have to #include them manually. (at least for CUDA 8.0)

__global__ void mykernel() 
{
	// map from threadIdx/BlockIdx to pixel position
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	const int offset = x + y * blockDim.x * gridDim.x;

	printf("mykernel at [%d, %d] offset=%d\n", x, y, offset);
}

extern"C" void 
main_cuda_print(int argc, char* argv[])
{
	int gpu_count = 0;
	hipError_t err = hipGetDeviceCount(&gpu_count);
	if(err)
	{
		printf("cudaGetDeviceCount() error = %d\n", err);
		return;
	}
	else
	{
		printf("gpu_count = %d\n", gpu_count);
	}
	
	mykernel<<<2, 3>>>();
}
