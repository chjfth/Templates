
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Note: nvcc.exe will include necessary CUDA .h headers automatically,
// so we do not have to #include them manually. (at least for CUDA 8.0)

extern"C" void 
main_cuda_print(int argc, char* argv[])
{
	int gpu_count = 0;
	hipError_t err = hipGetDeviceCount(&gpu_count);
	if(err)
	{
		printf("cudaGetDeviceCount() error = %d\n", err);
		return;
	}
	else
	{
		printf("gpu_count = %d\n", gpu_count);
	}
}
